#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#include "cuPrintf.cu"
#include "Function.cuh"

__global__ void testKernel()
{
	size_t size = 123;
	char* ptr = (char*)malloc(size);
	memset(ptr, 0, size);
	printf("Thread %d got pointer: %p\n", threadIdx.x, ptr);
	free(ptr);
}

__host__ void init()
{
	//cudaPrintfInit();

	testKernel<<<10, 64>>>();

	{
		hipError_t cudaerr = hipDeviceSynchronize();
		if (cudaerr != hipSuccess)
			printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));
		else
			printf("success.\n");
	}

	hipDeviceSynchronize();

	//hipDeviceSynchronize();
	//cudaPrintfDisplay(stdout, true);
	//cudaPrintfEnd();
}